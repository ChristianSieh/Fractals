#include "hip/hip_runtime.h"
/************************************************************************
  Program: Fractals
  Author: Charles Bonn and Christian Sieh
  Class: CSC433
  Instructor: John Weiss
  Date: 10/21/2016
  Description:  Mandelbrot functions
  Known bugs/missing features: N/A
************************************************************************/

#include "Mandel.h"


static GLint maxIter = 1000;

/* Calculate the square of a complex number. */
/************************************************************************
   Function: complexSquare
   Author: Taken from book
   Description: calculates the square of a complex number
   Parameters:
 complexNum z - a complex number of a point
 ************************************************************************/
__device__ complexNum complexSquare_para ( complexNum z )
{
    complexNum zSquare;
    zSquare.x = z.x * z.x - z.y * z.y;
    zSquare.y = 2 * z.x * z.y;
    return zSquare;
}

/* Calculate the square of a complex number. */
/************************************************************************
   Function: complexSquare
   Author: Taken from book
   Description: calculates the square of a complex number
   Parameters:
 complexNum z - a complex number of a point
 ************************************************************************/
complexNum complexSquare ( complexNum z )
{
    complexNum zSquare;
    zSquare.x = z.x * z.x - z.y * z.y;
    zSquare.y = 2 * z.x * z.y;
    return zSquare;
}

/************************************************************************
  Function: mandelSqTransf
  Author: Taken from book
  Description: squares complex values
  Parameters:
 complexNum z0 -
       Glint maxIter -
************************************************************************/
 GLint mandelSqTransf ( complexNum z0, GLint maxIter )
{
    complexNum z = z0;
    GLint count = 0;

    /* Quit when z * z > 4 */
    while ( ( z.x * z.x + z.y * z.y <= 4.0 ) && ( count < maxIter ) )
    {
	z = complexSquare( z );
        z.x += z0.x;
        z.y += z0.y;
        count++;
    }
    return count;
}

/************************************************************************
  Function: mandelSqTransf
  Author: Taken from book
  Description: squares complex values
  Parameters:
 complexNum z0 -
       Glint maxIter -
************************************************************************/
__global__ void  mandelSqTransf_para (   int maxIter , point *points )
{
    complexNum z, z0;
    GLint count = 0;

    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    z.x = points[i].x;
    z.y = points[i].y;
    z0.x = points[i].x;
    z0.y = points[i].y;

    /* Quit when z * z > 4 */
    while ( ( z.x * z.x + z.y * z.y <= 4.0 ) && ( count < maxIter ) )
    {
	    z = complexSquare_para ( z );
	    z.x += z0.x;
	    z.y += z0.y;
	    count++;
    }

     points[i].colorSpot = count;  
}
/************************************************************************
  Function: mandelbrot
  Author: Taken from book and modified by Charles Bonn and Christian Sieh
  Description: calculates mandelbrot points
  Parameters:
       GLint nx - x points
       Glint ny - y points 
       Glint maxIter - number of iterations per point
       vector<complexNum> &points - vector of points
************************************************************************/
void mandelbrot_para ( GLint nx, GLint ny, point *points, cX cmplx)
{
    complexNum z, zIncr;
  
    int i = 0;
    int size = nx * ny;

    
    //copy points

    /* initialize complex values */
    zIncr.x = cmplx.cW / ( GLfloat ( nx ) );
    zIncr.y = cmplx.cH / ( GLfloat ( ny ) );


    /* go though x values */
    for ( z.x = cmplx.xCMin; z.x < cmplx.xCMax; z.x += zIncr.x )
    {
        for ( z.y = cmplx.yCMin; z.y < cmplx.yCMax; z.y += zIncr.y )
        {
            /* Save point values to point */
            points[i].x = z.x;
            points[i].y = z.y;
	        i++;
         }	
    }

    point *d_points;

    hipMalloc( ( void ** )&d_points, ((size * 10) * sizeof(point)) );
    hipMemcpy( d_points, points, ((size * 10) * sizeof(point)) , hipMemcpyHostToDevice );

    int nThreads = 1024;
    int nBlocks = (10000000 + nThreads -1 ) / nThreads;

    mandelSqTransf_para<<< nBlocks, nThreads >>>(maxIter,d_points);	
    hipMemcpy( points, d_points, ((size * 10) * sizeof(point)) , hipMemcpyDeviceToHost );
    hipFree( d_points ); //free memory
}

/************************************************************************
  Function: mandelbrot
  Author: Taken from book and modified by Charles Bonn and Christian Sieh
  Description: calculates mandelbrot points
  Parameters:
 GLint nx - x points
       Glint ny - y points 
       Glint maxIter - number of iterations per point
       vector<complexNum> &points - vector of points
************************************************************************/
void mandelbrot ( GLint nx, GLint ny, point *points, cX cmplx)
{
    complexNum z, zIncr;
    point currPoint;
    int i = 0;
    int iterCount;

    /* initialize complex values */
    zIncr.x = cmplx.cW / ( GLfloat ( nx ) );
    zIncr.y = cmplx.cH / ( GLfloat ( ny ) );
	
    /* go though x values */
    for ( z.x = cmplx.xCMin; z.x < cmplx.xCMax; z.x += zIncr.x )
    {
	/* go though y values */
        for ( z.y = cmplx.yCMin; z.y < cmplx.yCMax; z.y += zIncr.y )
        {
	        /* Calculate point value */ 
            iterCount = mandelSqTransf ( z, maxIter );

	        /* Save point values to point */
	        points[i].x = z.x;
	        points[i].y = z.y;
	        points[i].colorSpot = iterCount;
               
		    /* push point to stack */
                
		    i++;
	    }
    }
}

/************************************************************************
  Function: mandelInit
  Author: takenfrom book and modified by Charles Bonn and Christian Sieh
  Description: init for mandelbrot points
  Parameters: vector<point> &points - vector of points
************************************************************************/
void mandelInit ( point *points , cX cmplx, bool parallel)
{
    /* Set number of x and y subdivisions and the max iterations. */
    GLint nx = 1000, ny = 1000;//, maxIter = 1500;
    glClear ( GL_COLOR_BUFFER_BIT );
    chrono::time_point<std::chrono::system_clock> c;
    

    /* Clear display window. */
    if( parallel == false )
    {
        c = chrono::system_clock::now();
        mandelbrot( nx, ny, points, cmplx);
	    chrono::duration<double> d_cpu = chrono::system_clock::now() - c;
    }
    else
    {
        c = chrono::system_clock::now();
	    mandelbrot_para(nx, ny, points, cmplx);
        chrono::duration<double> d_gpu = chrono::system_clock::now() - c;
    }
}

