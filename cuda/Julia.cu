#include "hip/hip_runtime.h"
 /************************************************************************
   Program: Fractals
   Author: Charles Bonn and Christian Sieh
   Class: CSC433
   Instructor: Dr. John Weiss
   Date: 10/21/2016
   Description: This file holds the functions used to compute the Julia Set
   Known bugs/missing features: N/A
 ************************************************************************/

#include "Julia.h"

static GLint maxIter = 1000;

 /************************************************************************
   Function: juliaComplexSquare
   Author: Taken from book
   Description: This function squares a complex number and returns it.
                This function is the parallel version.
   Parameters:
        complexNum z - The point that is used as z in z^2 + c
 ************************************************************************/
__device__ complexNum juliaComplexSquare_para ( complexNum z )
{
    complexNum zSquare;
    zSquare.x = z.x * z.x - z.y * z.y;
    zSquare.y = 2 * z.x * z.y;
    return zSquare;
}

 /************************************************************************
   Function: juliaComplexSquare
   Author: Taken from book
   Description: This function squares a complex number and returns it
   Parameters:
        complexNum z - The point that is used as z in z^2 + c
 ************************************************************************/
complexNum juliaComplexSquare ( complexNum z )
{
    complexNum zSquare;
    zSquare.x = z.x * z.x - z.y * z.y;
    zSquare.y = 2 * z.x * z.y;
    return zSquare;
}

 /************************************************************************
   Function: juliaSqTransf
   Author: Taken from book
   Description: This function computes z = z^2 + c repeatedly until z is
             greater than 4 or we reach maxIter. We then return count
             so we are able to tell how long it took z to diverge.
   Parameters:
        complexNum c - The initial point that is used as c in z^2 + c
        complexNum z - The point that is used as z in z^2 + c
        GLint maxIter - If count == maxIter then the point doesn't diverge
            and we return.
 ************************************************************************/
GLint juliaSqTransf ( complexNum c, complexNum z, GLint maxIter )
{
    GLint count = 0;

    /* Quit when z * z > 4 */
    while ( ( z.x * z.x + z.y * z.y <= 4.0 ) && ( count < maxIter ) )
    {
        z = juliaComplexSquare(z);
        z.x += c.x;
        z.y += c.y;
        count++;
    }

    return count;
}

 /************************************************************************
   Function: juliaSqTransf
   Author: Taken from book
   Description: This function computes z = z^2 + c repeatedly until z is
             greater than 4 or we reach maxIter. We then return count
             so we are able to tell how long it took z to diverge.
             This function is the parallel version.
   Parameters:
        complexNum c - The initial point that is used as c in z^2 + c
        complexNum z - The point that is used as z in z^2 + c
        GLint maxIter - If count == maxIter then the point doesn't diverge
            and we return.
 ************************************************************************/
__global__ void juliaSqTransf_para ( point *points, int maxIter, complexNum c )
{
    complexNum z;
    GLint count = 0;

    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    z.x = points[i].x;
    z.y = points[i].y;

    /* Quit when z * z > 4 */
    while ( ( z.x * z.x + z.y * z.y <= 4.0 ) && ( count < maxIter ) )
    {
        z = juliaComplexSquare_para(z);
        z.x += c.x;
        z.y += c.y;
        count++;
    }

    points[i].colorSpot = count;
}

 /************************************************************************
   Function: julia_para
   Author: Christian Sieh
   Description: This function goes through the complex plane a zIncr number
                of times in order to calculate the point for that pixel.
                The iterCount is how long it takes for the point to diverge
                and colorspot is used by Color.cpp to create a color map.
                This function is the parallel version.
   Parameters:
        GLint nx - The number of x points our points vector will have
        Glint ny - The number of y points our points vector will have
        GLint maxIter - The maximum number of times we will run juliaSqTransf
        vector<point> points - A vector that will hold our computed points
            for the Julia set 
        complexNum c - The initial point that is used as c in z^2 + c
 ************************************************************************/
void julia_para ( GLint nx, GLint ny, point *points, complexNum c, cX cmplx )
{
    complexNum z, zIncr;
    point currPoint;

    int i = 0;
    int size = nx * ny;

    /* initialize complex values */
    zIncr.x = cmplx.cW / ( GLfloat ( nx ) );
    zIncr.y = cmplx.cH / ( GLfloat ( ny ) );

    /* go though x values */
    for ( z.x = cmplx.xCMin; z.x < cmplx.xCMax; z.x += zIncr.x )
    {
        for ( z.y = cmplx.yCMin; z.y < cmplx.yCMax; z.y += zIncr.y )
        {
            /* Save point values to point */
            points[i].x = z.x;
            points[i].y = z.y;
	        i++;
        }
    }

    point *d_points;

    hipMalloc( ( void ** )&d_points, ((size * 10) * sizeof(point)) );
    hipMemcpy( d_points, points, ((size * 10) * sizeof(point)) , hipMemcpyHostToDevice );

    int nThreads = 1024;
    int nBlocks = (10000000 + nThreads -1 ) / nThreads; 

    juliaSqTransf_para<<< nBlocks, nThreads >>>(d_points, maxIter, c);	
    hipMemcpy( points, d_points, ((size * 10) * sizeof(point)) , hipMemcpyDeviceToHost );
    hipFree( d_points ); //free memory
}

 /************************************************************************
   Function: julia
   Author: Christian Sieh
   Description: This function goes through the complex plane a zIncr number
                of times in order to calculate the point for that pixel.
                The iterCount is how long it takes for the point to diverge
                and colorspot is used by Color.cpp to create a color map.
   Parameters:
        GLint nx - The number of x points our points vector will have
        Glint ny - The number of y points our points vector will have
        GLint maxIter - The maximum number of times we will run juliaSqTransf
        vector<point> points - A vector that will hold our computed points
            for the Julia set 
        complexNum c - The initial point that is used as c in z^2 + c
 ************************************************************************/
void julia ( GLint nx, GLint ny, point *points, complexNum c, cX cmplx )
{
    complexNum z, zIncr;
    point currPoint;
    int i = 0;
    GLint iterCount;

    /* initialize complex values */
    zIncr.x = cmplx.cW / ( GLfloat ( nx ) );
    zIncr.y = cmplx.cH / ( GLfloat ( ny ) );

    /* go though x values */
    for ( z.x = cmplx.xCMin; z.x < cmplx.xCMax; z.x += zIncr.x )
    {
	/* go though y values */
        for ( z.y = cmplx.yCMin; z.y < cmplx.yCMax; z.y += zIncr.y )
        {
	        /* Calculate point value */ 
            iterCount = juliaSqTransf ( c, z, maxIter );

	        /* Save point values to point */
	        currPoint.x = z.x;
	        currPoint.y = z.y;
	
	        currPoint.colorSpot = iterCount;

            i++;
        }
    }
}

 /************************************************************************
   Function: juliaInit
   Author: Christian Sieh
   Description: This function sets the number of iterations and points that
                we will generate in the complex plane.
   Parameters:
		vector<point> points - A vector that will hold our computed points
            for the Julia set 
		complexNum c - The initial point that is used as c in z^2 + c
 ************************************************************************/
void juliaInit(point *points, complexNum c, cX cmplx, bool parallel)
{
    /* Set number of x and y subdivisions and the max iterations. */
    GLint nx = 1000, ny = 1000;
    glClear(GL_COLOR_BUFFER_BIT); // Clear display window.
    chrono::time_point<std::chrono::system_clock> clock;

    /* Clear display window. */
    if( parallel == false )
    {
        clock = chrono::system_clock::now();
        julia(nx, ny, points, c, cmplx);
	    chrono::duration<double> d_cpu = chrono::system_clock::now() - clock;
    }
    else
    {
        clock = chrono::system_clock::now();
	    julia_para(nx, ny, points, c, cmplx);
        chrono::duration<double> d_gpu = chrono::system_clock::now() - clock;
    }
}
