/*
        ***** vecadd.cu *****

CUDA program to add two vectors.

Compile:    nvcc -o vecadd vecadd.cu
Usage:      vecadd [N], where N is vector length

Author: John M. Weiss, Ph.D.
CSC433/533 Computer Graphics - Fall 2016.

Modifications:
*/


#include <hip/hip_runtime.h>
#include <chrono>
#include <ctime>
#include <cmath>
#include <iostream>
using namespace std;


// sequential vector addition (on the host)
void add_seq( int *a, int *b, int *c, int n )
{
    for ( int i = 0; i < n; i++ )
        c[i] = a[i] + b[i];
}

// CUDA kernel: add two ints in parallel
__global__ void add_par( int *a, int *b, int *c, int n )
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if ( i < n ) c[i] = a[i] + b[i];
}

// generate array of random ints with values between 0 and n-1
void random_ints( int array[], int n )
{
    for ( int i = 0; i < n; i++ )
        array[i] = rand() % n;
}

int main( int argc, char** argv )
{
    int n = 1024;
    if ( argc > 1 ) n = atoi( argv[1] );

    // alloc host memory for vectors a, b, c
    int size = n * sizeof( int );
    int *a = ( int * )malloc( size );
    int *b = ( int * )malloc( size );
    int *c_seq = ( int * )malloc( size );
    int *c_par = ( int * )malloc( size );

    // fill arrays a and b with random ints
    srand( time( NULL ) );
    random_ints( a, n );
    random_ints( b, n );

    // add vectors sequentially
    auto c = chrono::system_clock::now();
    add_seq( a, b, c_seq, n );
    chrono::duration<double> d_cpu = chrono::system_clock::now() - c;

    // alloc device  memory for vectors a, b, c
    int *d_a, *d_b, *d_c;
    hipMalloc( ( void ** )&d_a, size );
    hipMalloc( ( void ** )&d_b, size );
    hipMalloc( ( void ** )&d_c, size );

    // copy vectors a and b to device
    hipMemcpy( d_a, a, size, hipMemcpyHostToDevice );
    hipMemcpy( d_b, b, size, hipMemcpyHostToDevice );

    // launch add() kernel on GPU with M threads per block, (N+M-1)/M blocks
    int nThreads = 64;                              // should be multiple of 32 (up to 1024)
    int nBlocks = ( n + nThreads - 1 ) / nThreads;
    c = chrono::system_clock::now();
    add_par<<< nBlocks, nThreads >>>(d_a, d_b, d_c, n);
    chrono::duration<double> d_gpu = chrono::system_clock::now() - c;

    // copy vector sum back to host
    hipMemcpy( c_par, d_c, size, hipMemcpyDeviceToHost );

    #if 0
    cout << "\na:    ";
    for ( int i = 0; i < n; i++ ) cout << " " << a[i];
    cout << "\nb:    ";
    for ( int i = 0; i < n; i++ ) cout << " " << b[i];
    cout << "\nc_seq:";
    for ( int i = 0; i < n; i++ ) cout << " " << c_seq[i];
    cout << "\nc_par:";
    for ( int i = 0; i < n; i++ ) cout << " " << c_par[i];
    #endif

    cout << "\n\nBenchmarks: CPU " << d_cpu.count() << " sec, GPU " << d_gpu.count() << " sec\n\n";

    // cleanup
    free( a ); free( b ); free( c_seq ); free( c_par );
    hipFree( d_a ); hipFree( d_b ); hipFree( d_c );
    return 0;
}
